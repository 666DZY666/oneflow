#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/kernel.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void SliceForwardGpu(const int64_t n, const int64_t* offset, const T* entire, T* slice) {
  CUDA_1D_KERNEL_LOOP(i, n) { slice[i] = entire[offset[i]]; }
}

template<typename T>
__global__ void SliceBackwardGpu(const int64_t n, const int64_t* offset, const T* slice,
                                 T* entire) {
  CUDA_1D_KERNEL_LOOP(i, n) { entire[offset[i]] = slice[i]; }
}

}  // namespace

template<typename T>
class SliceGpuKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(SliceGpuKernel);
  SliceGpuKernel() = default;
  ~SliceGpuKernel() = default;

 private:
  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    const Blob* in_blob = BnInOp2Blob("in");
    const Blob* offset_blob = BnInOp2Blob("out_to_in_offset");
    Blob* out_blob = BnInOp2Blob("out");
    const int64_t num_output = out_blob->shape().elem_cnt();
    SliceForwardGpu<T><<<BlocksNum4ThreadsNum(num_output), kCudaThreadsNumPerBlock, 0,
                         ctx.device_ctx->hip_stream()>>>(
        num_output, offset_blob->dptr<int64_t>(), in_blob->dptr<T>(), out_blob->mut_dptr<T>());
  }
  void InitConstBufBlobs(DeviceCtx* ctx,
                         std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    Shape in_shape(this->kernel_conf().slice_conf().in_shape());
    InitOut2InOffsetFromHost(ctx, in_shape, BnInOp2Blob("out_to_in_offset"));
  }

  void InitOut2InOffsetFromHost(DeviceCtx* ctx, const Shape& in_shape, Blob* blob) const {
    const SliceOpConf& conf = op_conf().slice_conf();
    WithHostBlobAndStreamSynchronizeEnv(ctx, blob, [&](Blob* host_blob) {
      int64_t* host_blob_ptr = host_blob->mut_dptr<int64_t>();
      FOR_RANGE(int64_t, i, 0, host_blob->shape().elem_cnt()) {
        int64_t offset = 0;
        int64_t index = i;
        FOR_RANGE(int64_t, j, 0, host_blob->shape().NumAxes()) {
          const int64_t dim_elem_cnt = host_blob->shape().Count(j + 1);
          const int64_t dim_i = index / dim_elem_cnt;
          index = index % dim_elem_cnt;
          int64_t start = 0;
          int64_t stride = 1;
          const DimSliceConf& dim_slice_conf = conf.dim_slice_conf(j);
          if (dim_slice_conf.has_start()) { start = dim_slice_conf.start(); }
          if (start < 0) { start += host_blob->shape().At(j); }
          stride = dim_slice_conf.stride();
          offset += (start + dim_i * stride) * in_shape.Count(j + 1);
        }
        host_blob_ptr[i] = offset;
      }
    });
  }
};

template<typename T>
class SliceGradGpuKernel final : public KernelIf<DeviceType::kGPU> {
 public:
  OF_DISALLOW_COPY_AND_MOVE(SliceGradGpuKernel);
  SliceGradGpuKernel() = default;
  ~SliceGradGpuKernel() = default;

 private:
  void ForwardDataContent(const KernelCtx& ctx,
                          std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    const Blob* dy_blob = BnInOp2Blob("dy");
    const Blob* offset_blob = BnInOp2Blob("y_to_x_offset");
    Blob* dx_blob = BnInOp2Blob("dx");
    const int64_t num_output = dy_blob->shape().elem_cnt();
    Memset<DeviceType::kGPU>(ctx.device_ctx, dx_blob->mut_dptr<T>(), 0,
                             dx_blob->ByteSizeOfBlobBody());
    SliceBackwardGpu<T><<<BlocksNum4ThreadsNum(num_output), kCudaThreadsNumPerBlock, 0,
                          ctx.device_ctx->hip_stream()>>>(
        num_output, offset_blob->dptr<int64_t>(), dy_blob->dptr<T>(), dx_blob->mut_dptr<T>());
  }
  void InitConstBufBlobs(DeviceCtx* ctx,
                         std::function<Blob*(const std::string&)> BnInOp2Blob) const override {
    Shape in_shape(this->kernel_conf().slice_conf().in_shape());
    InitOut2InOffsetFromHost(ctx, in_shape, BnInOp2Blob("y_to_x_offset"));
  }

  void InitOut2InOffsetFromHost(DeviceCtx* ctx, const Shape& in_shape, Blob* blob) const {
    const SliceGradOpConf& conf = op_conf().slice_grad_conf();
    WithHostBlobAndStreamSynchronizeEnv(ctx, blob, [&](Blob* host_blob) {
      int64_t* host_blob_ptr = host_blob->mut_dptr<int64_t>();
      FOR_RANGE(int64_t, i, 0, host_blob->shape().elem_cnt()) {
        int64_t offset = 0;
        int64_t index = i;
        FOR_RANGE(int64_t, j, 0, host_blob->shape().NumAxes()) {
          const int64_t dim_elem_cnt = host_blob->shape().Count(j + 1);
          const int64_t dim_i = index / dim_elem_cnt;
          index = index % dim_elem_cnt;
          int64_t start = 0;
          int64_t stride = 1;
          const DimSliceConf& dim_slice_conf = conf.dim_slice_conf(j);
          if (dim_slice_conf.has_start()) { start = dim_slice_conf.start(); }
          if (start < 0) { start += host_blob->shape().At(j); }
          stride = dim_slice_conf.stride();
          offset += (start + dim_i * stride) * in_shape.Count(j + 1);
        }
        host_blob_ptr[i] = offset;
      }
    });
  }
};

#define REGISTER_SLICE_GPU_KERNEL(dtype)                                                       \
  REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kSliceConf, DeviceType::kGPU, dtype,     \
                                        SliceGpuKernel<dtype>)                                 \
  REGISTER_KERNEL_WITH_DEVICE_AND_DTYPE(OperatorConf::kSliceGradConf, DeviceType::kGPU, dtype, \
                                        SliceGradGpuKernel<dtype>)

REGISTER_SLICE_GPU_KERNEL(float);
REGISTER_SLICE_GPU_KERNEL(double);
REGISTER_SLICE_GPU_KERNEL(int8_t);
REGISTER_SLICE_GPU_KERNEL(int32_t);
REGISTER_SLICE_GPU_KERNEL(int64_t);

}  // namespace oneflow
