#include "hip/hip_runtime.h"
#include "oneflow/core/kernel/one_hot_kernel.h"
#include "oneflow/core/kernel/kernel_util.cuh"
#include <assert.h>

namespace oneflow {

namespace {

template<typename T, typename K>
__global__ void OneHotEncodeGpu(int64_t elem_cnt, const K* indices, int64_t lower_bound,
    int64_t upper_bound, T* out) {
  const int64_t length = upper_bound - lower_bound;
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    const int64_t row = i / length;
    const int64_t col = i % length + lower_bound;
    const int64_t idx = indices[row];
    out[i] = (idx == col);
  }
}

}  // namespace

template<typename T, typename K>
struct OneHotKernelUtil<DeviceType::kGPU, T, K> final {
  static void Encode(DeviceCtx* ctx, const K* indices, int64_t num_indices, int64_t lower_bound,
      int64_t upper_bound, T* out);
};

template<typename T, typename K>
void OneHotKernelUtil<DeviceType::kGPU, T, K>::Encode(DeviceCtx* ctx, const K* indices,
                                                      int64_t num_indices, int64_t lower_bound,
                                                      int64_t upper_bound, T* out) {
  const int64_t elem_cnt = num_indices * (upper_bound - lower_bound);
  OneHotEncodeGpu<T, K>
      <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          elem_cnt, indices, lower_bound, upper_bound, out);
}

#define INSTANTIATE_ONE_HOT_KERNEL_UTIL_GPU(data_type_pair, index_type_pair)           \
  template struct OneHotKernelUtil<DeviceType::kGPU, OF_PP_PAIR_FIRST(data_type_pair), \
                                   OF_PP_PAIR_FIRST(index_type_pair)>;
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_ONE_HOT_KERNEL_UTIL_GPU, ARITHMETIC_DATA_TYPE_SEQ,
                                 INT_DATA_TYPE_SEQ);
#undef INSTANTIATE_ONE_HOT_KERNEL_UTIL_GPU

}  // namespace oneflow
