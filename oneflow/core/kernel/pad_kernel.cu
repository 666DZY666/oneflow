#include "hip/hip_runtime.h"
#include "oneflow/core/device/cuda_util.h"
#include "oneflow/core/kernel/kernel_util.h"
#include "oneflow/core/kernel/pad_kernel.h"
#include "oneflow/core/kernel/kernel_util.cuh"
#include <hipcub/hipcub.hpp>

namespace oneflow{
namespace{

template<typename T>
__global__ void PadOneAfter(const int64_t elem_cnt, const int64_t num_axes,
                            const int32_t* outshape_count,const int32_t* outshape_at,
                            const int32_t* inshape_count,const int32_t* inshape_at,
                            const T* in_dptr, T* out_dptr) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    int64_t offset = i;
    int64_t index = 0; 
    for(int64_t d = 0; d < num_axes; d++){
      int64_t dim = offset / outshape_count[d];
      // if this dim need padding
      if(dim >= inshape_at[d]){
        out_dptr[i] = ZeroVal<T>::value;
        break;
      }
      index += dim * inshape_count[d];
      offset -= dim * outshape_count[d];
      if(offset == 0){out_dptr[i] = in_dptr[index];}
    }
  }
}

template<typename T>
__global__ void PadOneAfterBackward(const int64_t elem_cnt, const int64_t num_axes,
                            const int32_t* outshape_count,const int32_t* outshape_at,
                            const int32_t* inshape_count,const int32_t* inshape_at,
                            T* in_diff_dptr, const T* out_diff_dptr) {
  CUDA_1D_KERNEL_LOOP(i, elem_cnt) {
    int64_t offset = i;
    int64_t index = 0; 
    for(int64_t d = 0; d < num_axes; d++){
      int64_t dim = offset / outshape_count[d];
      // if this dim need padding
      if(dim >= inshape_at[d]){break;}
      index += dim * inshape_count[d];
      offset -= dim * outshape_count[d];
      if(offset == 0){in_diff_dptr[index] = out_diff_dptr[i];}
    }
  }
}

}// namespace 

template<typename T>
struct PadKernelUtil<DeviceType::kGPU, T>{
  static void Forward(const KernelCtx& ctx, int32_t* outshape_count, int32_t* outshape_at,
                      int32_t* inshape_count, int32_t* inshape_at, 
                      const Blob* in_blob, Blob* out_blob){

    const Shape& outshape = out_blob->shape();
    const Shape& inshape = in_blob->shape();
    const int64_t elem_cnt = out_blob->shape().elem_cnt();
    int64_t num_axes = outshape.NumAxes();

    int32_t size = num_axes * sizeof(int32_t);
    int32_t h_outshape_count[num_axes];
    int32_t h_outshape_at[num_axes];
    int32_t h_inshape_at[num_axes];
    int32_t h_inshape_count[num_axes];

    for(int64_t i = 0; i < num_axes; i++){
      h_outshape_at[i] = static_cast<int32_t>(outshape.At(i));
      h_inshape_at[i] = static_cast<int32_t>(inshape.At(i));
      h_outshape_count[i] = static_cast<int32_t>(outshape.Count(i + 1));
      h_inshape_count[i] = static_cast<int32_t>(inshape.Count(i + 1));
    }

    CudaCheck(hipMemcpy(outshape_count, h_outshape_count, size, hipMemcpyHostToDevice));
    CudaCheck(hipMemcpy(outshape_at, h_outshape_at, size, hipMemcpyHostToDevice));
    CudaCheck(hipMemcpy(inshape_count, h_inshape_count, size, hipMemcpyHostToDevice));
    CudaCheck(hipMemcpy(inshape_at, h_inshape_at, size, hipMemcpyHostToDevice));

    PadOneAfter<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                ctx.device_ctx->hip_stream()>>>(elem_cnt, num_axes, outshape_count, outshape_at,
                inshape_count, inshape_at, in_blob->dptr<T>(), out_blob->mut_dptr<T>());
  }

  static void Backward(const KernelCtx& ctx, int32_t* outshape_count, int32_t* outshape_at,
                       int32_t* inshape_count, int32_t* inshape_at, 
                       Blob* in_diff_blob, const Blob* out_diff_blob) {
    const int64_t elem_cnt = out_diff_blob->shape().elem_cnt();
    int64_t num_axes = out_diff_blob->shape().NumAxes();
    
    PadOneAfterBackward<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                ctx.device_ctx->hip_stream()>>>(elem_cnt, num_axes, outshape_count, outshape_at,
                inshape_count, inshape_at, in_diff_blob->mut_dptr<T>(), out_diff_blob->dptr<T>());
  }
};

#define INSTANTIATE_PAD_KERNEL_UTIL(type_cpp, type_proto) \
  template class PadKernelUtil<DeviceType::kGPU, type_cpp>;
OF_PP_FOR_EACH_TUPLE(INSTANTIATE_PAD_KERNEL_UTIL, FLOATING_DATA_TYPE_SEQ);

}// namespace oneflow