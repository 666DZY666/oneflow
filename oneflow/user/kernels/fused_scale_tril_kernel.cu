#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/common/data_type.h"
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/util/cuda_half_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void FusedScaleTrilGpu(const int64_t elem_cnt, const int64_t num_rows,
                                  const int64_t num_cols, const int64_t diagonal, const T scale,
                                  const T* x, const T fill, T* y) {
  int64_t matrix_size = num_rows * num_cols;
  CUDA_1D_KERNEL_LOOP_T(int64_t, k, elem_cnt) {
    int64_t offset_in_matrix = k % matrix_size;
    int64_t i = offset_in_matrix / num_cols;
    int64_t j = offset_in_matrix - num_cols * i;
    y[k] = j > i + diagonal ? fill : (scale * x[k]);
  }
}

template<>
__global__ void FusedScaleTrilGpu<half>(const int64_t elem_cnt, const int64_t num_rows,
                                        const int64_t num_cols, const int64_t diagonal,
                                        const half scale, const half* x, const half fill, half* y) {
  int64_t matrix_size = num_rows * num_cols;
  const int64_t h2_n = elem_cnt / 2;
  half2 h2_scale = __half2half2(scale);
  const auto* x_h2 = reinterpret_cast<const half2*>(x);
  auto* y_h2 = reinterpret_cast<half2*>(y);
  CUDA_1D_KERNEL_LOOP_T(int64_t, k, h2_n) {
    half2 scale_x = __hmul2(h2_scale, x_h2[k]);
    int64_t offset_in_matrix = (2 * k) % matrix_size;
    int64_t i = offset_in_matrix / num_cols;
    int64_t j = offset_in_matrix - num_cols * i;
    half2 y_val;
    y_val.x = j > i + diagonal ? fill : scale_x.x;
    offset_in_matrix = (2 * k + 1) % matrix_size;
    i = offset_in_matrix / num_cols;
    j = offset_in_matrix - num_cols * i;
    y_val.y = j > i + diagonal ? fill : scale_x.y;
    y_h2[k] = y_val;
  }
  if (elem_cnt % 2 != 0 && blockIdx.x == 0 && threadIdx.x == 0) {
    const int64_t last_idx = elem_cnt - 1;
    int64_t offset_in_matrix = last_idx % matrix_size;
    int64_t i = offset_in_matrix / num_cols;
    int64_t j = offset_in_matrix - num_cols * i;
    y[last_idx] = j > i + diagonal ? fill : x[last_idx];
  }
}

__global__ void FusedScaleTrilGpuHalf2(const int64_t elem_cnt, const int64_t num_rows,
                                       const int64_t num_cols, const int64_t diagonal,
                                       const half scale, const half* x, const half fill, half* y) {
  const int64_t h2_n = elem_cnt / 2;
  const int64_t h2_num_cols = num_cols / 2;
  int64_t h2_matrix_size = num_rows * h2_num_cols;
  half2 h2_scale = __half2half2(scale);
  const auto* x_h2 = reinterpret_cast<const half2*>(x);
  auto* y_h2 = reinterpret_cast<half2*>(y);
  CUDA_1D_KERNEL_LOOP_T(int64_t, k, h2_n) {
    half2 scale_x = __hmul2(h2_scale, x_h2[k]);
    int64_t offset_in_h2_matrix = k % h2_matrix_size;
    int64_t i = offset_in_h2_matrix / h2_num_cols;
    int64_t j = offset_in_h2_matrix - h2_num_cols * i;
    half2 y_val;
    y_val.x = (2 * j) > i + diagonal ? fill : scale_x.x;
    y_val.y = (2 * j + 1) > i + diagonal ? fill : scale_x.y;
    y_h2[k] = y_val;
  }
}

template<typename T>
void FusedScaleTrilGpu(DeviceCtx* ctx, const int64_t elem_cnt, const int64_t num_rows,
                       const int64_t num_cols, const int64_t diagonal, const T scale, const T* x,
                       const T fill, T* y) {
  FusedScaleTrilGpu<T>
      <<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0, ctx->hip_stream()>>>(
          elem_cnt, num_rows, num_cols, diagonal, scale, x, fill, y);
}

template<>
void FusedScaleTrilGpu<half>(DeviceCtx* ctx, const int64_t elem_cnt, const int64_t num_rows,
                             const int64_t num_cols, const int64_t diagonal, const half scale,
                             const half* x, const half fill, half* y) {
  if (num_cols % 2 == 0) {
    FusedScaleTrilGpuHalf2<<<BlocksNum4ThreadsNum(elem_cnt), kCudaThreadsNumPerBlock, 0,
                             ctx->hip_stream()>>>(elem_cnt, num_rows, num_cols, diagonal, scale, x,
                                                   fill, y);
  } else {
    FusedScaleTrilGpu<half>(ctx, elem_cnt, num_rows, num_cols, diagonal, scale, x, fill, y);
  }
}

template<typename T>
T GetAttrVal(bool is_floating_val, double floating_value, int64_t integer_value) {
  return is_floating_val ? static_cast<T>(floating_value) : static_cast<T>(integer_value);
}

template<>
half GetAttrVal<half>(bool is_floating_val, double floating_value, int64_t integer_value) {
  return is_floating_val ? __float2half(floating_value) : __float2half(integer_value);
}

}  // namespace

template<typename T>
class GpuFusedScaleTrilKernel final : public user_op::OpKernel {
 public:
  GpuFusedScaleTrilKernel() = default;
  ~GpuFusedScaleTrilKernel() override = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx) const override {
    const user_op::Tensor* x = ctx->Tensor4ArgNameAndIndex("in", 0);
    const auto shape = x->shape();
    const auto diagonal = ctx->Attr<int64_t>("diagonal");
    const int64_t num_rows = shape.At(shape.NumAxes() - 2);
    const int64_t num_cols = shape.At(shape.NumAxes() - 1);
    user_op::Tensor* y = ctx->Tensor4ArgNameAndIndex("out", 0);
    const int64_t elem_cnt = shape.elem_cnt();
    const T fill = GetAttrVal<T>(ctx->Attr<bool>("is_floating_fill_value"),
                                 ctx->Attr<double>("floating_fill_value"),
                                 ctx->Attr<int64_t>("integer_fill_value"));
    const T scale = GetAttrVal<T>(ctx->Attr<bool>("is_floating_scale_value"),
                                  ctx->Attr<double>("floating_scale_value"),
                                  ctx->Attr<int64_t>("integer_scale_value"));
    FusedScaleTrilGpu<T>(ctx->device_ctx(), elem_cnt, num_rows, num_cols, diagonal, scale,
                         x->dptr<T>(), fill, y->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_GPU_FUSED_SCALE_TRIL_KERNEL(dtype)                                             \
  REGISTER_USER_KERNEL("fused_scale_tril")                                                      \
      .SetCreateFn<GpuFusedScaleTrilKernel<dtype>>()                                            \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                                       \
                       & (user_op::HobDataType("out", 0) == GetDataType<dtype>::value))         \
      .SetInplaceProposalFn([](const user_op::InferContext&,                                    \
                               user_op::AddInplaceArgPair AddInplaceArgPairFn) -> Maybe<void> { \
        OF_RETURN_IF_ERROR(AddInplaceArgPairFn("out", 0, "in", 0, true));                       \
        return Maybe<void>::Ok();                                                               \
      });

REGISTER_GPU_FUSED_SCALE_TRIL_KERNEL(float)
REGISTER_GPU_FUSED_SCALE_TRIL_KERNEL(double)
REGISTER_GPU_FUSED_SCALE_TRIL_KERNEL(int8_t)
REGISTER_GPU_FUSED_SCALE_TRIL_KERNEL(int32_t)
REGISTER_GPU_FUSED_SCALE_TRIL_KERNEL(int64_t)
REGISTER_GPU_FUSED_SCALE_TRIL_KERNEL(half)

}  // namespace oneflow
