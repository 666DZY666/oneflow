#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/common/balanced_splitter.h"
#include "oneflow/core/kernel/random_generator.h"
#include "oneflow/core/kernel/gather_kernel_util.h"
#include "oneflow/core/kernel/unsorted_segment_sum_kernel_util.h"
#ifdef WITH_CUDA
#include <hipcub/hipcub.hpp>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#endif

namespace oneflow {
namespace user_op {

namespace {

template<typename K>
int64_t GetCubSortPairTempStorageSize(int64_t n) {
  size_t cub_sort_temp_store_size = 0;
  OF_CUDA_CHECK((hipcub::DeviceRadixSort::SortPairs<K, K>(nullptr, cub_sort_temp_store_size, nullptr,
                                                       nullptr, nullptr, nullptr, n)));
  CHECK_GE(cub_sort_temp_store_size, 0);
  CHECK_LT(cub_sort_temp_store_size, GetMaxVal<int64_t>());
  return GetCudaAlignedSize(static_cast<int64_t>(cub_sort_temp_store_size));
}

template<typename KEY, typename VAL>
void SortPairs(hipStream_t stream, int64_t n, size_t temp_storage_bytes, const KEY* keys,
               const VAL* vals, void* tmp_storage, KEY* sorted_keys, VAL* sorted_vals) {
  OF_CUDA_CHECK((hipcub::DeviceRadixSort::SortPairs<KEY, VAL>(tmp_storage, temp_storage_bytes, keys,
                                                           sorted_keys, vals, sorted_vals, n, 0,
                                                           sizeof(KEY) * 8, stream)));
}

template<typename K>
class TmpBufferManager final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(TmpBufferManager);
  TmpBufferManager(void* ptr, const int64_t device_num_class) : ptr_(ptr) {
    const size_t label_buffer_bytes = GetCudaAlignedSize(device_num_class * sizeof(K));
    const size_t index_buffer_bytes = GetCudaAlignedSize(device_num_class * sizeof(K));
    const size_t sorted_label_buffer_bytes = GetCudaAlignedSize(device_num_class * sizeof(K));
    const size_t sorted_index_buffer_bytes = GetCudaAlignedSize(device_num_class * sizeof(K));
    const size_t rand_value_bytes = GetCudaAlignedSize(device_num_class * sizeof(unsigned int));
    cub_tmp_storage_bytes_ = GetCubSortPairTempStorageSize<K>(device_num_class);

    label_buffer_offset_ = 0;
    index_buffer_offset_ = label_buffer_offset_ + label_buffer_bytes;
    sorted_label_buffer_offset_ = index_buffer_offset_ + index_buffer_bytes;
    sorted_index_buffer_offset_ = sorted_label_buffer_offset_ + sorted_label_buffer_bytes;
    rand_value_offset_ = sorted_index_buffer_offset_ + sorted_index_buffer_bytes;
    cub_tmp_storage_offset_ = rand_value_offset_ + rand_value_bytes;
    total_buffer_size_ = label_buffer_bytes + index_buffer_bytes + sorted_label_buffer_bytes
                         + sorted_index_buffer_bytes + rand_value_bytes + cub_tmp_storage_bytes_;
  }
  ~TmpBufferManager() = default;

  size_t GetTotalBufferSize() const { return total_buffer_size_; }
  size_t GetCubTmpStorageSize() const { return cub_tmp_storage_bytes_; }
  K* LabelBufferPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + label_buffer_offset_);
  }
  K* IndexBufferPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + index_buffer_offset_);
  }
  K* SortedLabelBufferPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + sorted_label_buffer_offset_);
  }
  K* SortedIndexBufferPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + sorted_index_buffer_offset_);
  }
  unsigned int* RandValuePtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<unsigned int*>(reinterpret_cast<char*>(ptr_) + rand_value_offset_);
  }
  K* LabelMapPtr() const { return LabelBufferPtr(); }
  void* CubTmpStoragePtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<void*>(reinterpret_cast<char*>(ptr_) + cub_tmp_storage_offset_);
  }

 private:
  size_t label_buffer_offset_;
  size_t index_buffer_offset_;
  size_t sorted_label_buffer_offset_;
  size_t sorted_index_buffer_offset_;
  size_t rand_value_offset_;
  size_t cub_tmp_storage_offset_;
  size_t cub_tmp_storage_bytes_;
  size_t total_buffer_size_;
  void* ptr_;
};

int GetThreadNum(const hipDeviceProp_t& prop) {
  switch (prop.major) {
    case 3:  // Kepler
      return 2 * 192;
    case 5:  // Maxwell
      return 2 * 128;
    case 6:  // Pascal
      if ((prop.minor == 1) || (prop.minor == 2)) {
        return 2 * 128;
      } else {
        return 2 * 64;
      }
    case 7:  // Volta and Turing
      return 2 * 64;
    default: return 2 * 64;
  }
}

__global__ void SetupKernel(int64_t seed, hiprandState* state) {
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  size_t local_seed = (static_cast<size_t>(seed) + 0x9e3779b9U + (static_cast<size_t>(id) << 6U)
                       + (static_cast<size_t>(id) >> 2U));
  hiprand_init(local_seed, 0, 0, &state[id]);
}

template<typename K>
__global__ void GenerateGpu(hiprandState* state, const int64_t n, const int64_t max_val, K* buffer) {
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState localState = state[id];
  CUDA_1D_KERNEL_LOOP(i, n) { buffer[i] = static_cast<K>(hiprand(state) % max_val); }
  state[id] = localState;
}

class DistributedPartialFcSampleOpKernelState final : public user_op::OpKernelState {
 public:
  DistributedPartialFcSampleOpKernelState(DeviceCtx* ctx, int64_t lower, int64_t upper,
                                          int64_t num_sample_per_rank)
      : lower_(lower), upper_(upper), num_sample_per_rank_(num_sample_per_rank) {
    CHECK_NOTNULL(ctx);

    hipDeviceProp_t prop;
    OF_CUDA_CHECK(hipGetDeviceProperties(&prop, 0));
    block_num_ = prop.multiProcessorCount;
    thread_num_ = GetThreadNum(prop);
    OF_CUDA_CHECK(hipMalloc(&curand_states_, block_num_ * thread_num_ * sizeof(hiprandState)));
    SetupKernel<<<block_num_, thread_num_>>>(111L, curand_states_);
  }
  ~DistributedPartialFcSampleOpKernelState() { OF_CUDA_CHECK(hipFree(curand_states_)); };

  int64_t lower() const { return lower_; }
  int64_t upper() const { return upper_; }
  int64_t num_sample_per_rank() const { return num_sample_per_rank_; }

  template<typename K>
  void GenRandomIndexs(const int64_t n, const int64_t max_val, K* buffer) {
    GenerateGpu<K><<<block_num_, thread_num_>>>(curand_states_, n, max_val, buffer);
  }

 private:
  const int64_t lower_;
  const int64_t upper_;
  const int64_t num_sample_per_rank_;
  hiprandState* curand_states_;
  int32_t block_num_;
  int32_t thread_num_;
};

template<typename K>
__global__ void InitBuffer(const int64_t n, K* label_buffer) {
  CUDA_1D_KERNEL_LOOP(i, n) { label_buffer[i] = i; }
}

template<typename K>
__global__ void IndexSetPos(const int64_t n, const int64_t offset, const int64_t num_classes,
                            const K* labels, K* index_buffer) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    K label = labels[i] - offset;
    if (label >= 0 && label < num_classes) { index_buffer[label] = -1; }
  }
}

template<typename K>
__global__ void GetLabelMap(const int64_t n, const int64_t map_offset, const K* label,
                            K* label_map) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    K cur_label = label[i];
    label_map[cur_label] = map_offset + i;
  }
}

template<typename K>
__global__ void GetSampleLabel(const int64_t n, const int64_t offset, const K* label,
                               K* sample_label) {
  CUDA_1D_KERNEL_LOOP(i, n) { sample_label[i] = label[i] + offset; }
}

}  // namespace

template<typename T, typename K>
class DistributedPartialFcSampleGpuKernel final : public user_op::OpKernel {
 public:
  DistributedPartialFcSampleGpuKernel() = default;
  ~DistributedPartialFcSampleGpuKernel() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    const SbpParallel& in_sbp = ctx->SbpParallel4ArgNameAndIndex("weight", 0);
    const TensorDesc* in_logical_desc = ctx->LogicalTensorDesc4ArgNameAndIndex("weight", 0);
    const int64_t class_num = in_logical_desc->shape().At(0);
    const int64_t num_sample = ctx->Attr<int64_t>("num_sample");
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    const int64_t num_sample_per_rank = RoundUp(num_sample, parallel_num) / parallel_num;
    if (in_sbp.has_split_parallel() && in_sbp.split_parallel().axis() == 0 && parallel_num > 1) {
      CHECK(ctx->SbpParallel4ArgNameAndIndex("label", 0).has_broadcast_parallel());
      BalancedSplitter bs(class_num, parallel_num);
      return std::make_shared<DistributedPartialFcSampleOpKernelState>(
          ctx->device_ctx(), bs.At(ctx->parallel_ctx().parallel_id()).begin(),
          bs.At(ctx->parallel_ctx().parallel_id()).end(), num_sample_per_rank);
    } else {
      return std::make_shared<DistributedPartialFcSampleOpKernelState>(
          ctx->device_ctx(), 0, class_num, num_sample_per_rank);
    }
  }

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    const user_op::Tensor* weight = ctx->Tensor4ArgNameAndIndex("weight", 0);
    const user_op::Tensor* label = ctx->Tensor4ArgNameAndIndex("label", 0);
    user_op::Tensor* maped_label = ctx->Tensor4ArgNameAndIndex("maped_label", 0);
    user_op::Tensor* sampled_label = ctx->Tensor4ArgNameAndIndex("sampled_label", 0);
    user_op::Tensor* sampled_weight = ctx->Tensor4ArgNameAndIndex("sampled_weight", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);

    const int64_t batch_size = label->shape().At(0);
    const int64_t num_classes = weight->shape().At(0);
    TmpBufferManager<K> buffer_manager(tmp_buffer->mut_dptr(), num_classes);

    auto* kernel_state = dynamic_cast<DistributedPartialFcSampleOpKernelState*>(state);
    CHECK_NOTNULL(kernel_state);
    CHECK_EQ(weight->shape().At(0), kernel_state->upper() - kernel_state->lower());
    const int64_t upper_bound = kernel_state->upper();
    const int64_t lower_bound = kernel_state->lower();

    kernel_state->GenRandomIndexs<K>(num_classes, num_classes, buffer_manager.IndexBufferPtr());

    const int64_t num_sample = kernel_state->num_sample_per_rank();
    InitBuffer<<<BlocksNum4ThreadsNum(num_classes), kCudaThreadsNumPerBlock, 0,
                 ctx->device_ctx()->hip_stream()>>>(num_classes, buffer_manager.LabelBufferPtr());
    IndexSetPos<<<BlocksNum4ThreadsNum(batch_size), kCudaThreadsNumPerBlock, 0,
                  ctx->device_ctx()->hip_stream()>>>(
        batch_size, lower_bound, num_classes, label->dptr<K>(), buffer_manager.IndexBufferPtr());
    SortPairs<K, K>(ctx->device_ctx()->hip_stream(), num_classes,
                    buffer_manager.GetCubTmpStorageSize(), buffer_manager.IndexBufferPtr(),
                    buffer_manager.LabelBufferPtr(), buffer_manager.CubTmpStoragePtr(),
                    buffer_manager.SortedIndexBufferPtr(), buffer_manager.SortedLabelBufferPtr());
    // check num_sample > num_pos
    // get sampled_label
    GetSampleLabel<<<BlocksNum4ThreadsNum(num_sample), kCudaThreadsNumPerBlock, 0,
                     ctx->device_ctx()->hip_stream()>>>(num_sample, lower_bound,
                                                         buffer_manager.SortedLabelBufferPtr(),
                                                         sampled_label->mut_dptr<K>());
    // Memcpy<DeviceType::kGPU>(ctx->device_ctx(), sampled_label->mut_dptr<void>(),
    //                         buffer_manager.SortedLabelBufferPtr(),
    //                         num_sample * GetSizeOfDataType(sampled_label->data_type()));
    // get sampled weight
    GatherKernelUtilImpl<DeviceType::kGPU, T, K>::Forward(
        ctx->device_ctx(), buffer_manager.SortedLabelBufferPtr(), num_sample, weight->dptr<T>(),
        Shape({1, weight->shape().At(0), weight->shape().Count(1)}), sampled_weight->mut_dptr<T>(),
        0);
    // get LabelMap
    const int64_t map_offset = ctx->parallel_ctx().parallel_id() * num_sample;
    GetLabelMap<<<BlocksNum4ThreadsNum(num_sample), kCudaThreadsNumPerBlock, 0,
                  ctx->device_ctx()->hip_stream()>>>(num_sample, map_offset,
                                                      buffer_manager.SortedLabelBufferPtr(),
                                                      buffer_manager.LabelMapPtr());

    Memset<DeviceType::kGPU>(ctx->device_ctx(), maped_label->mut_dptr(), 0,
                             maped_label->shape().elem_cnt() * sizeof(K));
    GatherKernelUtilImpl<DeviceType::kGPU, K, K>::Forward(
        ctx->device_ctx(), label->dptr<K>(), batch_size, buffer_manager.LabelMapPtr(),
        Shape({1, num_classes, 1}), maped_label->mut_dptr<K>(), lower_bound);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_DISTRIBUTED_PARTIAL_FC_SAMPLE_GPU_KERNEL(dtype_pair, ltype_pair)                \
  REGISTER_USER_KERNEL("distributed_partial_fc_sample")                                          \
      .SetCreateFn<DistributedPartialFcSampleGpuKernel<OF_PP_PAIR_FIRST(dtype_pair),             \
                                                       OF_PP_PAIR_FIRST(ltype_pair)>>()          \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                                        \
                       & (user_op::HobDataType("label", 0) == OF_PP_PAIR_SECOND(ltype_pair))     \
                       & (user_op::HobDataType("weight", 0) == OF_PP_PAIR_SECOND(dtype_pair)))   \
      .SetInferTmpSizeFn([](oneflow::user_op::InferContext* ctx) {                               \
        const int64_t num_classes = ctx->TensorDesc4ArgNameAndIndex("weight", 0)->shape().At(0); \
        TmpBufferManager<OF_PP_PAIR_FIRST(ltype_pair)> buffer_manager(nullptr, num_classes);     \
        return buffer_manager.GetTotalBufferSize();                                              \
      });

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_DISTRIBUTED_PARTIAL_FC_SAMPLE_GPU_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

template<typename T, typename K>
class DistributedPartialFcSampleGradGpuKernel final : public user_op::OpKernel {
 public:
  DistributedPartialFcSampleGradGpuKernel() = default;
  ~DistributedPartialFcSampleGradGpuKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    const user_op::Tensor* sampled_weight_diff =
        ctx->Tensor4ArgNameAndIndex("sampled_weight_diff", 0);
    const user_op::Tensor* sampled_label = ctx->Tensor4ArgNameAndIndex("sampled_label", 0);
    user_op::Tensor* sampled_weight_diff_out =
        ctx->Tensor4ArgNameAndIndex("sampled_weight_diff_out", 0);
    user_op::Tensor* sampled_label_out = ctx->Tensor4ArgNameAndIndex("sampled_label_out", 0);
    Memcpy<DeviceType::kGPU>(ctx->device_ctx(), sampled_weight_diff_out->mut_dptr<void>(),
                             sampled_weight_diff->dptr<void>(),
                             sampled_weight_diff->shape().elem_cnt()
                                 * GetSizeOfDataType(sampled_weight_diff->data_type()));
    Memcpy<DeviceType::kGPU>(
        ctx->device_ctx(), sampled_label_out->mut_dptr<void>(), sampled_label->dptr<void>(),
        sampled_label->shape().elem_cnt() * GetSizeOfDataType(sampled_label->data_type()));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_DISTRIBUTED_PARTIAL_FC_SAMPLE_GRAD_GPU_KERNEL(dtype_pair, ltype_pair)      \
  REGISTER_USER_KERNEL("distributed_partial_fc_sample_grad")                                \
      .SetCreateFn<DistributedPartialFcSampleGradGpuKernel<OF_PP_PAIR_FIRST(dtype_pair),    \
                                                           OF_PP_PAIR_FIRST(ltype_pair)>>() \
      .SetIsMatchedHob(                                                                     \
          (user_op::HobDeviceTag() == "gpu")                                                \
          & (user_op::HobDataType("sampled_label", 0) == OF_PP_PAIR_SECOND(ltype_pair))     \
          & (user_op::HobDataType("sampled_weight_diff", 0) == OF_PP_PAIR_SECOND(dtype_pair)));
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_DISTRIBUTED_PARTIAL_FC_SAMPLE_GRAD_GPU_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

}  // namespace user_op
}  // namespace oneflow
