
#include <hip/hip_runtime.h>
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#ifdef WITH_CUDA

#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/common/balanced_splitter.h"
#include "oneflow/core/kernel/gather_kernel_util.h"
#include "oneflow/core/common/not_equal_to_previous_adjacent_iterator.h"
#include <cub/cub.cuh>
#include <curand.h>
#include <curand_kernel.h>

namespace oneflow {
namespace user_op {

namespace {

template<typename K>
int64_t GetCubSortPairsTempStorageSize(int64_t n) {
  size_t cub_sort_temp_store_size = 0;
  OF_CUDA_CHECK((cub::DeviceRadixSort::SortPairs<K, K>(nullptr, cub_sort_temp_store_size, nullptr,
                                                       nullptr, nullptr, nullptr, n)));
  size_t temp_store_size = GetCudaAlignedSize(static_cast<int64_t>(cub_sort_temp_store_size));
  CHECK_GE(temp_store_size, 0);
  CHECK_LT(temp_store_size, GetMaxVal<int64_t>());
  return temp_store_size;
}

template<typename K>
void CubSortPairs(cudaStream_t stream, int64_t n, size_t temp_storage_bytes, const K* keys,
                  const K* vals, void* tmp_storage, K* sorted_keys, K* sorted_vals) {
  OF_CUDA_CHECK(
      (cub::DeviceRadixSort::SortPairs<K, K>(tmp_storage, temp_storage_bytes, keys, sorted_keys,
                                             vals, sorted_vals, n, 0, sizeof(K) * 8, stream)));
}

template<typename K>
int64_t GetCubScanTempStorageSize(int64_t n) {
  size_t cub_scan_temp_store_size = 0;
  NotEqualToPreviousAdjacentIterator<K, K> unique_counting_iter(nullptr, 0);
  OF_CUDA_CHECK((cub::DeviceScan::InclusiveSum<NotEqualToPreviousAdjacentIterator<K, K>, K*>(
      nullptr, cub_scan_temp_store_size, unique_counting_iter, nullptr, n)));
  size_t temp_store_size = GetCudaAlignedSize(static_cast<int64_t>(cub_scan_temp_store_size));
  CHECK_GE(temp_store_size, 0);
  CHECK_LT(temp_store_size, GetMaxVal<int64_t>());
  return temp_store_size;
}

template<typename K>
class TmpBufferManager final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(TmpBufferManager);
  TmpBufferManager(void* ptr, const int64_t device_num_class, const int64_t batch_size)
      : ptr_(ptr) {
    const int64_t buffer_elem_cnt = std::max(device_num_class, batch_size);
    const size_t cub_sort_keys_bytes = GetCudaAlignedSize(buffer_elem_cnt * sizeof(K));
    const size_t cub_sort_values_bytes = GetCudaAlignedSize(buffer_elem_cnt * sizeof(K));
    const size_t cub_sort_keys_out_bytes = GetCudaAlignedSize(buffer_elem_cnt * sizeof(K));
    const size_t cub_sort_values_out_bytes = GetCudaAlignedSize(buffer_elem_cnt * sizeof(K));
    cub_tmp_storage_bytes_ = std::max(GetCubSortPairsTempStorageSize<K>(buffer_elem_cnt),
                                      GetCubScanTempStorageSize<K>(batch_size));
    cub_sort_keys_offset_ = 0;
    cub_sort_values_offset_ = cub_sort_keys_offset_ + cub_sort_keys_bytes;
    cub_sort_keys_out_offset_ = cub_sort_values_offset_ + cub_sort_keys_bytes;
    cub_sort_values_out_offset_ = cub_sort_keys_out_offset_ + cub_sort_keys_out_bytes;
    cub_tmp_storage_offset_ = cub_sort_values_out_offset_ + cub_sort_values_out_bytes;
    total_buffer_size_ = cub_sort_keys_bytes + cub_sort_values_bytes + cub_sort_keys_out_bytes
                         + cub_sort_values_out_bytes + cub_tmp_storage_bytes_;
  }
  ~TmpBufferManager() = default;

  size_t GetTotalBufferSize() const { return total_buffer_size_; }
  size_t GetCubTmpStorageSize() const { return cub_tmp_storage_bytes_; }
  K* CubSortKeysPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + cub_sort_keys_offset_);
  }
  K* CubSortValuesPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + cub_sort_values_offset_);
  }
  K* CubSortKeysOutPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + cub_sort_keys_out_offset_);
  }
  K* CubSortValuesOutPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + cub_sort_values_out_offset_);
  }
  void* CubTmpStoragePtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<void*>(reinterpret_cast<char*>(ptr_) + cub_tmp_storage_offset_);
  }

 private:
  size_t cub_sort_keys_offset_;
  size_t cub_sort_values_offset_;
  size_t cub_sort_keys_out_offset_;
  size_t cub_sort_values_out_offset_;
  size_t cub_tmp_storage_offset_;
  size_t cub_tmp_storage_bytes_;
  size_t total_buffer_size_;
  void* ptr_;
};

__global__ void SetupKernel(int64_t seed, curandState* state) {
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  size_t local_seed = (static_cast<size_t>(seed) + 0x9e3779b9U + (static_cast<size_t>(id) << 6U)
                       + (static_cast<size_t>(id) >> 2U));
  curand_init(local_seed, 0, 0, &state[id]);
}

template<typename K>
__global__ void GenerateGpu(curandState* state, const int64_t n, const int64_t max_val, K* buffer) {
  const int id = blockIdx.x * blockDim.x + threadIdx.x;
  curandState localState = state[id];
  CUDA_1D_KERNEL_LOOP(i, n) { buffer[i] = static_cast<K>(curand(state) % max_val); }
  state[id] = localState;
}

class DistributedPartialFcSampleOpKernelState final : public user_op::OpKernelState {
 public:
  DistributedPartialFcSampleOpKernelState(DeviceCtx* ctx, int64_t lower, int64_t upper,
                                          int64_t num_sample_per_rank, int64_t seed)
      : lower_(lower), upper_(upper), num_sample_per_rank_(num_sample_per_rank) {
    CHECK_NOTNULL(ctx);
    const int64_t num_classes = upper_ - lower_;
    OF_CUDA_CHECK(cudaMalloc(&curand_states_, BlocksNum4ThreadsNum(num_classes)
                                                  * kCudaThreadsNumPerBlock * sizeof(curandState)));
    SetupKernel<<<BlocksNum4ThreadsNum(num_classes), kCudaThreadsNumPerBlock, 0,
                  ctx->cuda_stream()>>>(seed, curand_states_);
  }
  ~DistributedPartialFcSampleOpKernelState() { OF_CUDA_CHECK(cudaFree(curand_states_)); };

  int64_t lower() const { return lower_; }
  int64_t upper() const { return upper_; }
  int64_t num_sample_per_rank() const { return num_sample_per_rank_; }

  template<typename K>
  void GenRandomIndexs(DeviceCtx* ctx, const int64_t n, const int64_t max_val, K* buffer) {
    GenerateGpu<K><<<BlocksNum4ThreadsNum(n), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
        curand_states_, n, max_val, buffer);
  }

 private:
  const int64_t lower_;
  const int64_t upper_;
  const int64_t num_sample_per_rank_;
  curandState* curand_states_;
};

template<typename K>
__global__ void IotaKernel(int64_t n, K* out) {
  CUDA_1D_KERNEL_LOOP(i, n) { out[i] = static_cast<K>(i); }
}

template<typename K>
__global__ void IndexSetPos(const int64_t n, const int64_t offset, const int64_t num_classes,
                            const K* labels, K* out) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    K label = labels[i] - offset;
    if (label >= 0 && label < num_classes) { out[label] = -1; }
  }
}

template<typename K>
__global__ void GetSampleLabel(const int64_t n, const int64_t offset, const K* label,
                               K* sample_label) {
  CUDA_1D_KERNEL_LOOP(i, n) { sample_label[i] = label[i] + offset; }
}

template<typename K>
__global__ void GetLabelMap(const int64_t n, const int64_t parallel_num,
                            const int64_t num_sample_per_rank, const K* bound_index,
                            const K* bound_value, K* label_map) {
  CUDA_1D_KERNEL_LOOP(i, n) {
#pragma unroll
    for (int64_t j = 0; j < parallel_num; j++) {
      if (i >= bound_index[j] && i < bound_index[j + 1]) {
        label_map[i] = label_map[i] - bound_value[j] + j * num_sample_per_rank;
      }
    }
  }
}

template<typename K>
__global__ void GetPartionBound(const int64_t n, const int64_t parallel_num,
                                const int64_t num_classes_per_rank, const K* key_ptr,
                                const K* value_ptr, K* bound_index, K* bound_value) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    if (i != 0 && i != n - 1) {
      const K cur_in = key_ptr[i];
      const K pre_in = key_ptr[i - 1];
#pragma unroll
      for (int32_t j = 1; j < parallel_num; ++j) {
        const int32_t lower_bound = j * num_classes_per_rank;
        if (cur_in >= lower_bound && pre_in < lower_bound) {
          bound_index[j] = static_cast<K>(i);
          bound_value[j] = value_ptr[i];
        }
      }
    }
    if (i == 0) {
      const K in = key_ptr[i];
#pragma unroll
      for (int32_t j = 0; j <= parallel_num; ++j) {
        const int32_t lower_bound = j * num_classes_per_rank;
        if (in >= lower_bound) {
          bound_index[j] = 0;
          bound_value[j] = value_ptr[i];
        }
      }
    }
    if (i == n - 1) {
      const K in = key_ptr[i];
#pragma unroll
      for (int32_t j = parallel_num; j >= 0; --j) {
        const int32_t lower_bound = j * num_classes_per_rank;
        if (in < lower_bound) {
          bound_index[j] = n;
          bound_value[j] = value_ptr[i];
        }
      }
    }
  }
}

template<typename K>
__global__ void GetMappedLabel(const int64_t n, const K* label_map_key, const K* label_map_value,
                               K* maped_label) {
  CUDA_1D_KERNEL_LOOP(i, n) { maped_label[label_map_key[i]] = label_map_value[i]; }
}

template<typename K>
void SampleIndex(DeviceCtx* ctx, const int64_t num_classes, const int64_t batch_size,
                 const int64_t lower_bound, const TmpBufferManager<K>& buffer_manager,
                 const K* label_ptr) {
  IotaKernel<<<BlocksNum4ThreadsNum(num_classes), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
      num_classes, buffer_manager.CubSortValuesPtr());
  IndexSetPos<<<BlocksNum4ThreadsNum(batch_size), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
      batch_size, lower_bound, num_classes, label_ptr, buffer_manager.CubSortKeysPtr());
  CubSortPairs<K>(ctx->cuda_stream(), num_classes, buffer_manager.GetCubTmpStorageSize(),
                  buffer_manager.CubSortKeysPtr(), buffer_manager.CubSortValuesPtr(),
                  buffer_manager.CubTmpStoragePtr(), buffer_manager.CubSortKeysOutPtr(),
                  buffer_manager.CubSortValuesOutPtr());
}

template<typename K>
void MapLabel(DeviceCtx* ctx, const int64_t num_classes, const int64_t batch_size,
              const int64_t lower_bound, const int64_t parallel_num, const int64_t num_sample,
              const TmpBufferManager<K>& buffer_manager, const K* label_ptr, K* maped_label_ptr) {
  IotaKernel<<<BlocksNum4ThreadsNum(batch_size), kCudaThreadsNumPerBlock, 0, ctx->cuda_stream()>>>(
      batch_size, buffer_manager.CubSortValuesPtr());
  CubSortPairs<K>(ctx->cuda_stream(), batch_size, buffer_manager.GetCubTmpStorageSize(), label_ptr,
                  buffer_manager.CubSortValuesPtr(), buffer_manager.CubTmpStoragePtr(),
                  buffer_manager.CubSortKeysOutPtr(), buffer_manager.CubSortValuesOutPtr());

  size_t temp_storage_bytes = buffer_manager.GetCubTmpStorageSize();
  NotEqualToPreviousAdjacentIterator<K, K> unique_counting_iter(buffer_manager.CubSortKeysOutPtr(),
                                                                0);
  OF_CUDA_CHECK((cub::DeviceScan::InclusiveSum<NotEqualToPreviousAdjacentIterator<K, K>, K*>(
      buffer_manager.CubTmpStoragePtr(), temp_storage_bytes, unique_counting_iter,
      buffer_manager.CubSortValuesPtr(), batch_size, ctx->cuda_stream())));

  K* bound_index = buffer_manager.CubSortKeysPtr();
  K* bound_value = buffer_manager.CubSortKeysPtr() + parallel_num + 1;
  GetPartionBound<<<BlocksNum4ThreadsNum(batch_size), kCudaThreadsNumPerBlock, 0,
                    ctx->cuda_stream()>>>(
      batch_size, parallel_num, num_classes, buffer_manager.CubSortKeysOutPtr(),
      buffer_manager.CubSortValuesPtr(), bound_index, bound_value);

  GetLabelMap<K>
      <<<BlocksNum4ThreadsNum(batch_size), kCudaThreadsNumPerBlock, parallel_num * sizeof(K),
         ctx->cuda_stream()>>>(batch_size, parallel_num, num_sample, bound_index, bound_value,
                               buffer_manager.CubSortValuesPtr());

  GetMappedLabel<<<BlocksNum4ThreadsNum(batch_size), kCudaThreadsNumPerBlock, 0,
                   ctx->cuda_stream()>>>(batch_size, buffer_manager.CubSortValuesOutPtr(),
                                         buffer_manager.CubSortValuesPtr(), maped_label_ptr);
}

}  // namespace

template<typename T, typename K>
class DistributedPartialFcSampleGpuKernel final : public user_op::OpKernel {
 public:
  DistributedPartialFcSampleGpuKernel() = default;
  ~DistributedPartialFcSampleGpuKernel() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    const SbpParallel& in_sbp = ctx->SbpParallel4ArgNameAndIndex("weight", 0);
    const TensorDesc* in_logical_desc = ctx->LogicalTensorDesc4ArgNameAndIndex("weight", 0);
    const int64_t class_num = in_logical_desc->shape().At(0);
    const int64_t num_sample = ctx->Attr<int64_t>("num_sample");
    const int64_t seed = ctx->Attr<int64_t>("seed");
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    const int64_t num_sample_per_rank = RoundUp(num_sample, parallel_num) / parallel_num;
    if (in_sbp.has_split_parallel() && in_sbp.split_parallel().axis() == 0 && parallel_num > 1) {
      CHECK(ctx->SbpParallel4ArgNameAndIndex("label", 0).has_broadcast_parallel());
      BalancedSplitter bs(class_num, parallel_num);
      return std::make_shared<DistributedPartialFcSampleOpKernelState>(
          ctx->device_ctx(), bs.At(ctx->parallel_ctx().parallel_id()).begin(),
          bs.At(ctx->parallel_ctx().parallel_id()).end(), num_sample_per_rank, seed);
    } else {
      return std::make_shared<DistributedPartialFcSampleOpKernelState>(
          ctx->device_ctx(), 0, class_num, num_sample_per_rank, seed);
    }
  }

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    const user_op::Tensor* weight = ctx->Tensor4ArgNameAndIndex("weight", 0);
    const user_op::Tensor* label = ctx->Tensor4ArgNameAndIndex("label", 0);
    user_op::Tensor* maped_label = ctx->Tensor4ArgNameAndIndex("maped_label", 0);
    user_op::Tensor* sampled_label = ctx->Tensor4ArgNameAndIndex("sampled_label", 0);
    user_op::Tensor* sampled_weight = ctx->Tensor4ArgNameAndIndex("sampled_weight", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);

    const int64_t batch_size = label->shape().At(0);
    const int64_t num_classes = weight->shape().At(0);
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    TmpBufferManager<K> buffer_manager(tmp_buffer->mut_dptr(), num_classes, batch_size);

    auto* kernel_state = dynamic_cast<DistributedPartialFcSampleOpKernelState*>(state);
    CHECK_NOTNULL(kernel_state);
    CHECK_EQ(num_classes, kernel_state->upper() - kernel_state->lower());
    const int64_t lower_bound = kernel_state->lower();
    const int64_t num_sample = kernel_state->num_sample_per_rank();
    kernel_state->GenRandomIndexs<K>(ctx->device_ctx(), num_classes, num_classes,
                                     buffer_manager.CubSortKeysPtr());
    SampleIndex<K>(ctx->device_ctx(), num_classes, batch_size, lower_bound, buffer_manager,
                   label->dptr<K>());

    GetSampleLabel<<<BlocksNum4ThreadsNum(num_sample), kCudaThreadsNumPerBlock, 0,
                     ctx->device_ctx()->cuda_stream()>>>(num_sample, lower_bound,
                                                         buffer_manager.CubSortValuesOutPtr(),
                                                         sampled_label->mut_dptr<K>());

    GatherKernelUtilImpl<DeviceType::kGPU, T, K>::Forward(
        ctx->device_ctx(), buffer_manager.CubSortValuesOutPtr(), num_sample, weight->dptr<T>(),
        Shape({1, weight->shape().At(0), weight->shape().Count(1)}), sampled_weight->mut_dptr<T>(),
        0);

    MapLabel<K>(ctx->device_ctx(), num_classes, batch_size, lower_bound, parallel_num, num_sample,
                buffer_manager, label->dptr<K>(), maped_label->mut_dptr<K>());
  }

  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_DISTRIBUTED_PARTIAL_FC_SAMPLE_GPU_KERNEL(dtype_pair, ltype_pair)                \
  REGISTER_USER_KERNEL("distributed_partial_fc_sample")                                          \
      .SetCreateFn<DistributedPartialFcSampleGpuKernel<OF_PP_PAIR_FIRST(dtype_pair),             \
                                                       OF_PP_PAIR_FIRST(ltype_pair)>>()          \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                                        \
                       & (user_op::HobDataType("label", 0) == OF_PP_PAIR_SECOND(ltype_pair))     \
                       & (user_op::HobDataType("weight", 0) == OF_PP_PAIR_SECOND(dtype_pair)))   \
      .SetInferTmpSizeFn([](oneflow::user_op::InferContext* ctx) {                               \
        const int64_t num_classes = ctx->TensorDesc4ArgNameAndIndex("weight", 0)->shape().At(0); \
        const int64_t batch_size = ctx->TensorDesc4ArgNameAndIndex("label", 0)->shape().At(0);   \
        TmpBufferManager<OF_PP_PAIR_FIRST(ltype_pair)> buffer_manager(nullptr, num_classes,      \
                                                                      batch_size);               \
        return buffer_manager.GetTotalBufferSize();                                              \
      });

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_DISTRIBUTED_PARTIAL_FC_SAMPLE_GPU_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

template<typename T, typename K>
class DistributedPartialFcSampleGradGpuKernel final : public user_op::OpKernel {
 public:
  DistributedPartialFcSampleGradGpuKernel() = default;
  ~DistributedPartialFcSampleGradGpuKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    const user_op::Tensor* sampled_weight_diff =
        ctx->Tensor4ArgNameAndIndex("sampled_weight_diff", 0);
    const user_op::Tensor* sampled_label = ctx->Tensor4ArgNameAndIndex("sampled_label", 0);
    user_op::Tensor* sampled_weight_diff_out =
        ctx->Tensor4ArgNameAndIndex("sampled_weight_diff_out", 0);
    user_op::Tensor* sampled_label_out = ctx->Tensor4ArgNameAndIndex("sampled_label_out", 0);
    Memcpy<DeviceType::kGPU>(ctx->device_ctx(), sampled_weight_diff_out->mut_dptr<void>(),
                             sampled_weight_diff->dptr<void>(),
                             sampled_weight_diff->shape().elem_cnt()
                                 * GetSizeOfDataType(sampled_weight_diff->data_type()));
    Memcpy<DeviceType::kGPU>(
        ctx->device_ctx(), sampled_label_out->mut_dptr<void>(), sampled_label->dptr<void>(),
        sampled_label->shape().elem_cnt() * GetSizeOfDataType(sampled_label->data_type()));
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_DISTRIBUTED_PARTIAL_FC_SAMPLE_GRAD_GPU_KERNEL(dtype_pair, ltype_pair)      \
  REGISTER_USER_KERNEL("distributed_partial_fc_sample_grad")                                \
      .SetCreateFn<DistributedPartialFcSampleGradGpuKernel<OF_PP_PAIR_FIRST(dtype_pair),    \
                                                           OF_PP_PAIR_FIRST(ltype_pair)>>() \
      .SetIsMatchedHob(                                                                     \
          (user_op::HobDeviceTag() == "gpu")                                                \
          & (user_op::HobDataType("sampled_label", 0) == OF_PP_PAIR_SECOND(ltype_pair))     \
          & (user_op::HobDataType("sampled_weight_diff", 0) == OF_PP_PAIR_SECOND(dtype_pair)));
OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_DISTRIBUTED_PARTIAL_FC_SAMPLE_GRAD_GPU_KERNEL,
                                 FLOATING_DATA_TYPE_SEQ, INDEX_DATA_TYPE_SEQ)

}  // namespace user_op
}  // namespace oneflow
#endif
