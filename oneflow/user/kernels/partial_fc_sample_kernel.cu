#include "hip/hip_runtime.h"
/*
Copyright 2020 The OneFlow Authors. All rights reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
*/
#include "oneflow/core/framework/framework.h"
#include "oneflow/core/kernel/new_kernel_util.h"
#include "oneflow/core/common/balanced_splitter.h"
#include "oneflow/core/kernel/random_generator.h"
#include "oneflow/core/kernel/gather_kernel_util.h"
#include "oneflow/core/kernel/unsorted_segment_sum_kernel_util.h"
#include <hipcub/hipcub.hpp>

namespace oneflow {
namespace user_op {

namespace {

template<typename K>
int64_t GetCubSortPairTempStorageSize(int64_t n) {
  size_t cub_sort_temp_store_size = 0;
  OF_CUDA_CHECK((hipcub::DeviceRadixSort::SortPairs<K, K>(nullptr, cub_sort_temp_store_size, nullptr,
                                                       nullptr, nullptr, nullptr, n)));
  CHECK_GE(cub_sort_temp_store_size, 0);
  CHECK_LT(cub_sort_temp_store_size, GetMaxVal<int64_t>());
  return GetCudaAlignedSize(static_cast<int64_t>(cub_sort_temp_store_size));
}

template<typename KEY, typename VAL>
void SortPairs(hipStream_t stream, int64_t n, size_t temp_storage_bytes, const KEY* keys,
               const VAL* vals, void* tmp_storage, KEY* sorted_keys, VAL* sorted_vals) {
  OF_CUDA_CHECK((hipcub::DeviceRadixSort::SortPairs<KEY, VAL>(tmp_storage, temp_storage_bytes, keys,
                                                           sorted_keys, vals, sorted_vals, n, 0,
                                                           sizeof(KEY) * 8, stream)));
}

template<typename K>
class TmpBufferManager final {
 public:
  OF_DISALLOW_COPY_AND_MOVE(TmpBufferManager);
  TmpBufferManager(void* ptr, const int64_t device_num_class) : ptr_(ptr) {
    const size_t label_buffer_bytes = GetCudaAlignedSize(device_num_class * sizeof(K));
    const size_t index_buffer_bytes = GetCudaAlignedSize(device_num_class * sizeof(K));
    const size_t sorted_label_buffer_bytes = GetCudaAlignedSize(device_num_class * sizeof(K));
    const size_t sorted_index_buffer_bytes = GetCudaAlignedSize(device_num_class * sizeof(K));
    const size_t rand_value_bytes = GetCudaAlignedSize(device_num_class * sizeof(unsigned int));
    cub_tmp_storage_bytes_ = GetCubSortPairTempStorageSize<K>(device_num_class);

    label_buffer_offset_ = 0;
    index_buffer_offset_ = label_buffer_offset_ + label_buffer_bytes;
    sorted_label_buffer_offset_ = index_buffer_offset_ + index_buffer_bytes;
    sorted_index_buffer_offset_ = sorted_label_buffer_offset_ + sorted_label_buffer_offset_;
    rand_value_offset_ = sorted_index_buffer_offset_ + sorted_index_buffer_bytes;
    cub_tmp_storage_offset_ = rand_value_offset_ + rand_value_bytes;
    total_buffer_size_ = label_buffer_bytes + index_buffer_bytes + sorted_label_buffer_bytes
                         + sorted_index_buffer_bytes + rand_value_bytes + cub_tmp_storage_bytes_;
  }
  ~TmpBufferManager() = default;

  size_t GetTotalBufferSize() const { return total_buffer_size_; }
  size_t GetCubTmpStorageSize() const { return cub_tmp_storage_bytes_; }
  K* LabelBufferPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + label_buffer_offset_);
  }
  K* IndexBufferPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + index_buffer_offset_);
  }
  K* SortedLabelBufferPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + sorted_label_buffer_offset_);
  }
  K* SortedIndexBufferPtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<K*>(reinterpret_cast<char*>(ptr_) + sorted_index_buffer_offset_);
  }
  unsigned int* RandValuePtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<unsigned int*>(reinterpret_cast<char*>(ptr_) + rand_value_offset_);
  }
  K* LabelMapPtr() const { return LabelBufferPtr(); }
  void* CubTmpStoragePtr() const {
    CHECK(ptr_ != nullptr);
    return reinterpret_cast<void*>(reinterpret_cast<char*>(ptr_) + cub_tmp_storage_offset_);
  }

 private:
  size_t label_buffer_offset_;
  size_t index_buffer_offset_;
  size_t sorted_label_buffer_offset_;
  size_t sorted_index_buffer_offset_;
  size_t rand_value_offset_;
  size_t cub_tmp_storage_offset_;
  size_t cub_tmp_storage_bytes_;
  size_t total_buffer_size_;
  void* ptr_;
};

class PartialFcSampleOpKernelState final : public user_op::OpKernelState {
 public:
  PartialFcSampleOpKernelState(DeviceCtx* ctx, int64_t lower, int64_t upper,
                               int64_t num_sample_per_rank)
      : lower_(lower), upper_(upper), num_sample_per_rank_(num_sample_per_rank) {
    CHECK_NOTNULL(ctx);
    OF_CURAND_CHECK(hiprandCreateGenerator(&curand_generator_, HIPRAND_RNG_PSEUDO_DEFAULT));
    OF_CURAND_CHECK(
        hiprandSetPseudoRandomGeneratorSeed(curand_generator_, static_cast<int64_t>(1111L)));
    OF_CURAND_CHECK(hiprandSetStream(curand_generator_, ctx->hip_stream()));
  }
  ~PartialFcSampleOpKernelState() override = default;

  int64_t lower() const { return lower_; }
  int64_t upper() const { return upper_; }
  int64_t num_sample_per_rank() const { return num_sample_per_rank_; }
  hiprandGenerator_t& gen() { return curand_generator_; }

 private:
  const int64_t lower_;
  const int64_t upper_;
  const int64_t num_sample_per_rank_;
  hiprandGenerator_t curand_generator_;
};

template<typename K>
__global__ void InitBuffer(const int64_t n, const unsigned int* rand_value, K* label_buffer,
                           K* index_buffer) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    label_buffer[i] = i;
    index_buffer[i] = static_cast<K>(rand_value[i] % n);
  }
}

template<typename K>
__global__ void IndexSetPos(const int64_t n, const int64_t offset, const int64_t num_classes,
                            const K* labels, K* index_buffer) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    K label = labels[i] - offset;
    if (label >= 0 && label < num_classes) { index_buffer[label] = -1; }
  }
}

template<typename K>
__global__ void GetLabelMap(const int64_t n, const int64_t map_offset, const K* label,
                            K* label_map) {
  CUDA_1D_KERNEL_LOOP(i, n) {
    K cur_label = label[i];
    label_map[cur_label] = map_offset + i;
  }
}

template<typename K>
__global__ void GetSampleLabel(const int64_t n, const int64_t offset, const K* label,
                               K* sample_label) {
  CUDA_1D_KERNEL_LOOP(i, n) { sample_label[i] = label[i] + offset; }
}

}  // namespace

template<typename T, typename K>
class PartialFcSampleGpuKernel final : public user_op::OpKernel {
 public:
  PartialFcSampleGpuKernel() = default;
  ~PartialFcSampleGpuKernel() = default;

  std::shared_ptr<user_op::OpKernelState> CreateOpKernelState(
      user_op::KernelInitContext* ctx) const override {
    const SbpParallel& in_sbp = ctx->SbpParallel4ArgNameAndIndex("weight", 0);
    const TensorDesc* in_logical_desc = ctx->LogicalTensorDesc4ArgNameAndIndex("weight", 0);
    const int64_t class_num = in_logical_desc->shape().At(0);
    const int64_t num_sample = ctx->Attr<int64_t>("num_sample");
    const int64_t parallel_num = ctx->parallel_ctx().parallel_num();
    const int64_t num_sample_per_rank = RoundUp(num_sample, parallel_num) / parallel_num;
    if (in_sbp.has_split_parallel() && in_sbp.split_parallel().axis() == 0 && parallel_num > 1) {
      CHECK(ctx->SbpParallel4ArgNameAndIndex("label", 0).has_broadcast_parallel());
      BalancedSplitter bs(class_num, parallel_num);
      return std::make_shared<PartialFcSampleOpKernelState>(
          ctx->device_ctx(), bs.At(ctx->parallel_ctx().parallel_id()).begin(),
          bs.At(ctx->parallel_ctx().parallel_id()).end(), num_sample_per_rank);
    } else {
      return std::make_shared<PartialFcSampleOpKernelState>(ctx->device_ctx(), 0, class_num,
                                                            num_sample_per_rank);
    }
  }

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    const user_op::Tensor* weight = ctx->Tensor4ArgNameAndIndex("weight", 0);
    const user_op::Tensor* label = ctx->Tensor4ArgNameAndIndex("label", 0);
    user_op::Tensor* maped_label = ctx->Tensor4ArgNameAndIndex("maped_label", 0);
    user_op::Tensor* sampled_label = ctx->Tensor4ArgNameAndIndex("sampled_label", 0);
    user_op::Tensor* sampled_weight = ctx->Tensor4ArgNameAndIndex("sampled_weight", 0);
    user_op::Tensor* tmp_buffer = ctx->Tensor4ArgNameAndIndex("tmp_buffer", 0);

    const int64_t batch_size = label->shape().At(0);
    const int64_t num_classes = weight->shape().At(0);
    TmpBufferManager<K> buffer_manager(tmp_buffer->mut_dptr(), num_classes);

    auto* kernel_state = dynamic_cast<PartialFcSampleOpKernelState*>(state);
    CHECK_NOTNULL(kernel_state);
    CHECK_EQ(weight->shape().At(0), kernel_state->upper() - kernel_state->lower());
    const int64_t upper_bound = kernel_state->upper();
    const int64_t lower_bound = kernel_state->lower();
    OF_CURAND_CHECK(
        hiprandGenerate(kernel_state->gen(), buffer_manager.RandValuePtr(), num_classes));
    const int64_t num_sample = kernel_state->num_sample_per_rank();
    InitBuffer<<<BlocksNum4ThreadsNum(num_classes), kCudaThreadsNumPerBlock, 0,
                 ctx->device_ctx()->hip_stream()>>>(num_classes, buffer_manager.RandValuePtr(),
                                                     buffer_manager.LabelBufferPtr(),
                                                     buffer_manager.IndexBufferPtr());
    IndexSetPos<<<BlocksNum4ThreadsNum(batch_size), kCudaThreadsNumPerBlock, 0,
                  ctx->device_ctx()->hip_stream()>>>(
        batch_size, lower_bound, num_classes, label->dptr<K>(), buffer_manager.IndexBufferPtr());
    SortPairs<K, K>(ctx->device_ctx()->hip_stream(), num_classes,
                    buffer_manager.GetCubTmpStorageSize(), buffer_manager.IndexBufferPtr(),
                    buffer_manager.LabelBufferPtr(), buffer_manager.CubTmpStoragePtr(),
                    buffer_manager.SortedIndexBufferPtr(), buffer_manager.SortedLabelBufferPtr());
    // check num_sample > num_pos
    // get sampled_label
    const bool indexed_slice_update = ctx->Attr<bool>("indexed_slice_update");
    if (indexed_slice_update) {
      GetSampleLabel<<<BlocksNum4ThreadsNum(num_sample), kCudaThreadsNumPerBlock, 0,
                       ctx->device_ctx()->hip_stream()>>>(num_sample, lower_bound,
                                                           buffer_manager.SortedLabelBufferPtr(),
                                                           sampled_label->mut_dptr<K>());
    } else {
      Memcpy<DeviceType::kGPU>(ctx->device_ctx(), sampled_label->mut_dptr<void>(),
                               buffer_manager.SortedLabelBufferPtr(),
                               num_sample * GetSizeOfDataType(sampled_label->data_type()));
    }
    // get sampled weight
    GatherKernelUtilImpl<DeviceType::kGPU, T, K>::Forward(
        ctx->device_ctx(), buffer_manager.SortedLabelBufferPtr(), num_sample, weight->dptr<T>(),
        Shape({1, weight->shape().At(0), weight->shape().Count(1)}), sampled_weight->mut_dptr<T>(),
        0);
    // get LabelMap
    const int64_t map_offset = ctx->parallel_ctx().parallel_id() * num_sample;
    GetLabelMap<<<BlocksNum4ThreadsNum(num_sample), kCudaThreadsNumPerBlock, 0,
                  ctx->device_ctx()->hip_stream()>>>(num_sample, map_offset,
                                                      buffer_manager.SortedLabelBufferPtr(),
                                                      buffer_manager.LabelMapPtr());

    Memset<DeviceType::kGPU>(ctx->device_ctx(), maped_label->mut_dptr(), 0,
                             maped_label->shape().elem_cnt() * sizeof(K));
    GatherKernelUtilImpl<DeviceType::kGPU, K, K>::Forward(
        ctx->device_ctx(), label->dptr<K>(), batch_size, buffer_manager.LabelMapPtr(),
        Shape({1, num_classes, 1}), maped_label->mut_dptr<K>(), lower_bound);
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_PARTIAL_FC_SAMPLE_GPU_KERNEL(dtype_pair, ltype_pair)                             \
  REGISTER_USER_KERNEL("partial_fc_sample")                                                       \
      .SetCreateFn<                                                                               \
          PartialFcSampleGpuKernel<OF_PP_PAIR_FIRST(dtype_pair), OF_PP_PAIR_FIRST(ltype_pair)>>() \
      .SetIsMatchedHob((user_op::HobDeviceTag() == "gpu")                                         \
                       & (user_op::HobDataType("label", 0) == OF_PP_PAIR_SECOND(ltype_pair))      \
                       & (user_op::HobDataType("weight", 0) == OF_PP_PAIR_SECOND(dtype_pair)))    \
      .SetInferTmpSizeFn([](oneflow::user_op::InferContext* ctx) {                                \
        const int64_t num_classes = ctx->TensorDesc4ArgNameAndIndex("weight", 0)->shape().At(0);  \
        TmpBufferManager<OF_PP_PAIR_FIRST(ltype_pair)> buffer_manager(nullptr, num_classes);      \
        return buffer_manager.GetTotalBufferSize();                                               \
      });

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_PARTIAL_FC_SAMPLE_GPU_KERNEL, FLOATING_DATA_TYPE_SEQ,
                                 INDEX_DATA_TYPE_SEQ)

template<typename T, typename K>
class PartialFcSampleGradGpuKernel final : public user_op::OpKernel {
 public:
  PartialFcSampleGradGpuKernel() = default;
  ~PartialFcSampleGradGpuKernel() = default;

 private:
  void Compute(user_op::KernelComputeContext* ctx, user_op::OpKernelState* state) const override {
    const user_op::Tensor* sampled_label = ctx->Tensor4ArgNameAndIndex("sampled_label", 0);
    const user_op::Tensor* sampled_weight_diff =
        ctx->Tensor4ArgNameAndIndex("sampled_weight_diff", 0);
    user_op::Tensor* weight_diff = ctx->Tensor4ArgNameAndIndex("weight_diff", 0);
    Memset<DeviceType::kGPU>(ctx->device_ctx(), weight_diff->mut_dptr(), 0,
                             weight_diff->shape().elem_cnt() * sizeof(T));
    UnsortedSegmentSumKernelUtil<DeviceType::kGPU, T, K>::UnsortedSegmentSum(
        ctx->device_ctx(), sampled_label->dptr<K>(), sampled_weight_diff->dptr<T>(),
        sampled_label->shape().elem_cnt(), weight_diff->shape().At(0), 1,
        weight_diff->shape().Count(1), 0, weight_diff->mut_dptr<T>());
  }
  bool AlwaysComputeWhenAllOutputsEmpty() const override { return false; }
};

#define REGISTER_PARTIAL_FC_SAMPLE_GRAD_GPU_KERNEL(dtype_pair, ltype_pair)              \
  REGISTER_USER_KERNEL("partial_fc_sample_grad")                                        \
      .SetCreateFn<PartialFcSampleGradGpuKernel<OF_PP_PAIR_FIRST(dtype_pair),           \
                                                OF_PP_PAIR_FIRST(ltype_pair)>>()        \
      .SetIsMatchedHob(                                                                 \
          (user_op::HobDeviceTag() == "gpu")                                            \
          & (user_op::HobDataType("sampled_label", 0) == OF_PP_PAIR_SECOND(ltype_pair)) \
          & (user_op::HobDataType("sampled_weight_diff", 0) == OF_PP_PAIR_SECOND(dtype_pair)));

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_PARTIAL_FC_SAMPLE_GRAD_GPU_KERNEL, FLOATING_DATA_TYPE_SEQ,
                                 INDEX_DATA_TYPE_SEQ)

}  // namespace user_op
}  // namespace oneflow
