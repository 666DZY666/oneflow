#include "hip/hip_runtime.h"
#include "oneflow/customized/kernels/clip_by_value_kernel.h"
#include "oneflow/core/kernel/util/cuda_kernel_util.h"

namespace oneflow {

namespace {

template<typename T>
__global__ void CudaClipByMin(int64_t num_values, const T* values, const T* min_value, T* out_ptr) {
  ClipValuesByMin<DeviceType::kGPU>(num_values, values, *min_value, out_ptr);
}

template<typename T>
__global__ void CudaClipByMax(int64_t num_values, const T* values, const T* max_value, T* out_ptr) {
  ClipValuesByMax<DeviceType::kGPU>(num_values, values, *max_value, out_ptr);
}

template<typename T>
__global__ void CudaClipByMinMax(int64_t num_values, const T* values, const T* min_value,
                                 const T* max_value, T* out_ptr) {
  ClipValuesByMinMax<DeviceType::kGPU>(num_values, values, *min_value, *max_value, out_ptr);
}

}  // namespace

template<typename T>
struct ClipFunctor<DeviceType::kGPU, T> {
  __device__ __forceinline__ static T Min(const T value, const T min_value) {
    return min(value, min_value);
  }
  __device__ __forceinline__ static T Max(const T value, const T max_value) {
    return max(value, max_value);
  }
};

template<typename T>
struct ClipValuesUtil<DeviceType::kGPU, T> {
  static void ByMin(DeviceCtx* ctx, int64_t num_values, const T* values, const T* min_value,
                    T* out_ptr) {
    RUN_CUDA_KERNEL((CudaClipByMin<T>), ctx, num_values, num_values, values, min_value, out_ptr);
  }

  static void ByMax(DeviceCtx* ctx, int64_t num_values, const T* values, const T* max_value,
                    T* out_ptr) {
    RUN_CUDA_KERNEL((CudaClipByMax<T>), ctx, num_values, num_values, values, max_value, out_ptr);
  }

  static void ByMinMax(DeviceCtx* ctx, int64_t num_values, const T* values, const T* min_value,
                       const T* max_value, T* out_ptr) {
    RUN_CUDA_KERNEL((CudaClipByMinMax<T>), ctx, num_values, num_values, values, min_value,
                    max_value, out_ptr);
  }
};

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(INSTANTIATE_CLIP_VALUES_UTIL, (DeviceType::kGPU),
                                 ARITHMETIC_DATA_TYPE_SEQ)

OF_PP_SEQ_PRODUCT_FOR_EACH_TUPLE(REGISTER_CLIP_BY_VALUE_KERNEL, (DeviceType::kGPU),
                                 ARITHMETIC_DATA_TYPE_SEQ)

}  // namespace oneflow
